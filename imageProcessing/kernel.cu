#include "hip/hip_runtime.h"
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

#include <iostream>

#include "hip/hip_runtime.h"
#include ""

using IMG = decltype(stbi_load(std::declval<const char*>(), std::declval<int*>(), std::declval<int*>(), std::declval<int*>(), std::declval<int>()));
constexpr int BLOCK = 16;


__global__ void blurImage(const IMG image, IMG out_image,  int width, int height, int channels) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int halfWindowSize = 10;

    if (x < width && y < height) {
        int idx = (y * width + x) * channels;
        int sum_r = 0;
        int sum_g = 0;
        int sum_b = 0;
        for (int i = -halfWindowSize; i <= halfWindowSize; i++) { // iterate over the 5x5 kernel
            for (int j = -halfWindowSize; j <= halfWindowSize; j++) { // iterate over the 5x5 kernel
                int x1 = x + i; // x coordinate of the kernel
                int y1 = y + j; // y coordinate of the kernel
                if (x1 >= 0 && x1 < width && y1 >= 0 && y1 < height) { // check if the kernel is within the image
                    int idx1 = (y1 * width + x1) * channels; // index of the kernel pixel

                    // sum the colors
                    sum_r += image[idx1];
                    sum_g += image[idx1 + 1];
                    sum_b += image[idx1 + 2];
                }
            }
        }

        // average the colors
        out_image[idx] = sum_r / (halfWindowSize * 2 + 1) / (halfWindowSize * 2 + 1);
        out_image[idx + 1] = sum_g / (halfWindowSize * 2 + 1) / (halfWindowSize * 2 + 1);
        out_image[idx + 2] = sum_b / (halfWindowSize * 2 + 1) / (halfWindowSize * 2 + 1);
    }
}

// rotate image
__global__ void rotateImage(const IMG image, IMG out_image, int width, int height, int channels, int random) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int idx = (y * width + x) * channels;
        int new_idx = 0;
        if (random % 3 == 0) {
            new_idx = (x * height + (height - y - 1)) * channels;
        }
        else if (random % 3 == 1) {
            new_idx = ((width - x - 1) * height + y) * channels;
        }
        else if (random % 3 == 2) {
            new_idx = ((height - y - 1) * width + (width - x - 1)) * channels;
        }
        else {
            new_idx = (y * width + x) * channels;
        }
        out_image[new_idx] = image[idx];
        out_image[new_idx + 1] = image[idx + 1];
        out_image[new_idx + 2] = image[idx + 2];
    }
}

__global__ void negativeImage(const IMG image, IMG out_image, int width, int height, int channels) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int idx = (y * width + x) * channels;
        out_image[idx] = 255 - image[idx];
        out_image[idx + 1] = 255 - image[idx + 1];
        out_image[idx + 2] = 255 - image[idx + 2];
    }
}


void assertCudaSuccess(hipError_t code) {
    if (code != hipSuccess) {
        std::cout << "Error: " << hipGetErrorString(code) << std::endl;
        exit(code);
    }
}

IMG readImageFromFile(const std::string& filename, int *width, int *height, int *channels)
{
    IMG image = stbi_load(filename.c_str(), width, height, channels, 0);
    if (!image) {
        throw std::runtime_error(stbi_failure_reason());
    }
    if (*width % 32 != 0 || *height % 32 != 0) {
        throw std::runtime_error("Image dimensions must be multiples of 32");
    }
    return image;
}


int main()
{
    srand(time(NULL));
    int width, height, channels;
    IMG image = readImageFromFile("img.jpg", &width, &height, &channels);

    IMG gpu_image, out_image;
    assertCudaSuccess(hipMalloc(&gpu_image, width * height * channels * sizeof(unsigned char)));
    assertCudaSuccess(hipMalloc(&out_image, width * height * channels * sizeof(unsigned char)));

    assertCudaSuccess(hipMemcpy(gpu_image, image, width * height * channels * sizeof(unsigned char), hipMemcpyHostToDevice));

    dim3 block(BLOCK, BLOCK); // 16x16 block
    dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y); // grid dimensions based on the image size and block size (rounded down)

     imageToGrayscale << <grid, block >> > (gpu_image, out_image, width, height, channels);
     assertCudaSuccess(hipMemcpy(image, out_image, width * height * channels * sizeof(unsigned char), hipMemcpyDeviceToHost));
     stbi_write_jpg("img_gray.jpg", width, height, channels, image, 100);

    blurImage << <grid, block >> > (gpu_image, out_image, width, height, channels);
    assertCudaSuccess(hipMemcpy(image, out_image, width * height * channels * sizeof(unsigned char), hipMemcpyDeviceToHost));
    stbi_write_jpg("img_blur.jpg", width, height, channels, image, 100);

    rotateImage << <grid, block >> > (gpu_image, out_image, width, height, channels, rand()*1000);
    assertCudaSuccess(hipMemcpy(image, out_image, width * height * channels * sizeof(unsigned char), hipMemcpyDeviceToHost));
    stbi_write_jpg("img_rotated.jpg", height, width, channels, image, 100);

    negativeImage << <grid, block >> > (gpu_image, out_image, width, height, channels);
    assertCudaSuccess(hipMemcpy(image, out_image, width * height * channels * sizeof(unsigned char), hipMemcpyDeviceToHost));
    stbi_write_jpg("img_negative.jpg", width, height, channels, image, 100);


    // Free memory
    stbi_image_free(image);
    hipFree(gpu_image);

    return 0;
}
